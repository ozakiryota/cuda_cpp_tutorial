
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld()
{
    printf("Hello World!\n");
}

int main()
{
    helloWorld<<<1, 3>>>();
    hipDeviceSynchronize();
}

/* OUTPUT

Hello World!
Hello World!
Hello World!

*/