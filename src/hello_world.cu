
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld()
{
    printf("Hello World!\n");
}

int main()
{
    size_t num_blocks = 1;
    size_t num_threads_per_block = 3;

    helloWorld<<<num_blocks, num_threads_per_block>>>();
    hipDeviceSynchronize();
}

/* OUTPUT

Hello World!
Hello World!
Hello World!

*/