
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printArray(int* arr, size_t num_elements)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    int grid_stride = gridDim.x * blockDim.x;

    for(int i = index; i < num_elements; i += grid_stride){
	    printf("arr[%d] = %d\n", i, arr[i]);
    }
}

int main()
{
    const size_t num_blocks = 2;
    const size_t num_threads_per_block = 3;
    const size_t num_elements = 10;

    int* arr;
    size_t bytes = num_elements * sizeof(int);
    hipMallocManaged(&arr, bytes);

    for(size_t i = 0; i < num_elements; i++)   arr[i] = i;

    printArray<<<num_blocks, num_threads_per_block>>>(arr, num_elements);
    hipDeviceSynchronize();
}

/* OUTPUT

arr[0] = 0
arr[1] = 1
arr[2] = 2
arr[3] = 3
arr[4] = 4
arr[5] = 5
arr[6] = 6
arr[7] = 7
arr[8] = 8
arr[9] = 9

*/