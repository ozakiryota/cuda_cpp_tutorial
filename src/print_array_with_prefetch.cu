#include "hip/hip_runtime.h"
#include <stdio.h>

void printArrayCPU(int* arr, size_t num_elements)
{
    for(size_t i = 0; i < num_elements; i++){
	    printf("arr[%d] = %d\n", i, arr[i]);
    }
}

__global__ void printArray(int* arr, size_t num_elements)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("arr[%d] = %d\n", index, arr[index]);
}

int main()
{
    const size_t num_blocks = 2;
    const size_t num_threads_per_block = 3;
    const size_t num_elements = num_blocks * num_threads_per_block;

    int* arr;
    size_t bytes = num_elements * sizeof(int);
    hipMallocManaged(&arr, bytes);

    for(size_t i = 0; i < num_elements; i++)   arr[i] = i;

    int device_id;
	hipGetDevice(&device_id);
    hipMemPrefetchAsync(arr, bytes, device_id);

	printf("----- GPU -----\n");
    printArray<<<num_blocks, num_threads_per_block>>>(arr, num_elements);
    hipDeviceSynchronize();

    printf("----- CPU -----\n");
    hipMemPrefetchAsync(arr, bytes, hipCpuDeviceId);
    printArrayCPU(arr, num_elements);
}

/* PROFILE

$ nsys profile --stats=true ~/cuda_cpp_tutorial/build/print_array_with_prefetch

----- GPU -----
arr[0] = 0
arr[1] = 1
arr[2] = 2
arr[3] = 3
arr[4] = 4
arr[5] = 5
----- CPU -----
arr[0] = 0
arr[1] = 1
arr[2] = 2
arr[3] = 3
arr[4] = 4
arr[5] = 5
Generating '/tmp/nsys-report-d472.qdstrm'
[1/8] [========================100%] report1.nsys-rep
[2/8] [========================100%] report1.sqlite
[3/8] Executing 'nvtxsum' stats report
SKIPPED: /root/cuda_cpp_tutorial/build/report1.sqlite does not contain NV Tools Extension (NVTX) data.
[4/8] Executing 'osrtsum' stats report

Operating System Runtime API Statistics:

 Time (%)  Total Time (ns)  Num Calls   Avg (ns)    Med (ns)   Min (ns)  Max (ns)  StdDev (ns)       Name     
 --------  ---------------  ---------  ----------  ----------  --------  --------  -----------  --------------
     70.2        231463209         17  13615482.9  10060535.0      3250  68769206   17963544.0  poll          
     14.2         46707197        454    102879.3     11345.0      1010  13064068     743131.1  ioctl         
      9.6         31819233         13   2447633.3     26640.0     10670  20429791    6081350.0  sem_timedwait 
      5.5         18114990         30    603833.0      3790.0      1210  17910430    3268710.9  fopen         
      0.2           752665         27     27876.5      3110.0      2150    466242      88468.7  mmap64        
      0.1           389781         44      8858.7      8125.5      3300     25340       4368.9  open64        
      0.1           176071          5     35214.2     42150.0     18800     45871      11728.4  pthread_create
      0.0           111910         18      6217.2      5510.0      1270     28480       6326.0  mmap          
      0.0            41450          1     41450.0     41450.0     41450     41450          0.0  fgets         
      0.0            32010          6      5335.0      5040.0      2450      7900       2152.6  open          
      0.0            24480          7      3497.1      3190.0      2390      5690       1155.1  munmap        
      0.0            23310         11      2119.1      2250.0      1090      3960        893.8  write         
      0.0            20590          9      2287.8      1250.0      1070      6800       2079.4  fcntl         
      0.0            19100          9      2122.2      1710.0      1080      3980       1060.0  fclose        
      0.0            17040          8      2130.0      1935.0      1350      3320        663.7  read          
      0.0            15720          2      7860.0      7860.0      3500     12220       6166.0  socket        
      0.0            13250          2      6625.0      6625.0      5820      7430       1138.4  fread         
      0.0            11230          2      5615.0      5615.0      1420      9810       5932.6  fwrite        
      0.0             9390          1      9390.0      9390.0      9390      9390          0.0  connect       
      0.0             8610          2      4305.0      4305.0      1040      7570       4617.4  fflush        
      0.0             7480          1      7480.0      7480.0      7480      7480          0.0  pipe2         
      0.0             2380          1      2380.0      2380.0      2380      2380          0.0  bind          

[5/8] Executing 'cudaapisum' stats report

CUDA API Statistics:

 Time (%)  Total Time (ns)  Num Calls   Avg (ns)     Med (ns)    Min (ns)   Max (ns)   StdDev (ns)          Name         
 --------  ---------------  ---------  -----------  -----------  ---------  ---------  -----------  ---------------------
     99.8        190180906          1  190180906.0  190180906.0  190180906  190180906          0.0  hipMallocManaged    
      0.1           216510          2     108255.0     108255.0      21790     194720     122280.0  hipMemPrefetchAsync 
      0.0            56570          1      56570.0      56570.0      56570      56570          0.0  hipDeviceSynchronize
      0.0            46641          1      46641.0      46641.0      46641      46641          0.0  hipLaunchKernel     

[6/8] Executing 'gpukernsum' stats report

CUDA Kernel Statistics:

 Time (%)  Total Time (ns)  Instances  Avg (ns)  Med (ns)  Min (ns)  Max (ns)  StdDev (ns)                Name              
 --------  ---------------  ---------  --------  --------  --------  --------  -----------  --------------------------------
    100.0            43168          1   43168.0   43168.0     43168     43168          0.0  printArray(int *, unsigned long)

[7/8] Executing 'gpumemtimesum' stats report

CUDA Memory Operation Statistics (by time):

 Time (%)  Total Time (ns)  Count  Avg (ns)  Med (ns)  Min (ns)  Max (ns)  StdDev (ns)              Operation            
 --------  ---------------  -----  --------  --------  --------  --------  -----------  ---------------------------------
     61.1             1856      1    1856.0    1856.0      1856      1856          0.0  [CUDA Unified Memory memcpy HtoD]
     38.9             1184      1    1184.0    1184.0      1184      1184          0.0  [CUDA Unified Memory memcpy DtoH]

[8/8] Executing 'gpumemsizesum' stats report

CUDA Memory Operation Statistics (by size):

 Total (MB)  Count  Avg (MB)  Med (MB)  Min (MB)  Max (MB)  StdDev (MB)              Operation            
 ----------  -----  --------  --------  --------  --------  -----------  ---------------------------------
      0.004      1     0.004     0.004     0.004     0.004        0.000  [CUDA Unified Memory memcpy DtoH]
      0.004      1     0.004     0.004     0.004     0.004        0.000  [CUDA Unified Memory memcpy HtoD]

Generated:
    /root/cuda_cpp_tutorial/build/report1.nsys-rep
    /root/cuda_cpp_tutorial/build/report1.sqlite

*/