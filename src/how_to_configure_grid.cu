
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hoge()
{
    int hoge = 1 + 1;
}

int main()
{
    /*device query*/
	int device_id;
	hipGetDevice(&device_id);
	int num_sm;
	hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, device_id);

    printf("num_sm = %d\n", num_sm);

    /*Grid sizes that are multiples of the number of available SMs can increase performance*/
    const size_t num_blocks = 32 * num_sm;
    const size_t num_threads_per_block = 256;

    hoge<<<num_blocks, num_threads_per_block>>>();
    hipDeviceSynchronize();
}

/* OUTPUT

num_sm = 28

*/