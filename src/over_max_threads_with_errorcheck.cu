
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printIndicies()
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("index = %d\n", index);
}

int main()
{
    const size_t num_blocks = 1;
    const size_t num_threads_per_block = 1025; // > 1024

    printIndicies<<<num_blocks, num_threads_per_block>>>();
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if(err != hipSuccess)  printf("Error: %s\n", hipGetErrorString(err));
}

/* OUTPUT

Error: invalid configuration argument

*/