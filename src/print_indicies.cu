
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printIndicies()
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("index = %d\n", index);
}

int main()
{
    printIndicies<<<2, 3>>>();
    hipDeviceSynchronize();
}

/* OUTPUT

index = 0
index = 1
index = 2
index = 3
index = 4
index = 5

*/