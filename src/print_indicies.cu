
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printIndicies()
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("index = %d\n", index);
}

int main()
{
    const size_t num_blocks = 2;
    const size_t num_threads_per_block = 3;

    printIndicies<<<num_blocks, num_threads_per_block>>>();
    hipDeviceSynchronize();
}

/* OUTPUT

index = 0
index = 1
index = 2
index = 3
index = 4
index = 5

*/