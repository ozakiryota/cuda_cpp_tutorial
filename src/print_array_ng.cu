
#include <hip/hip_runtime.h>
#include <stdio.h>

void printArrayCPU(int* arr, size_t num_elements)
{
    for(size_t i = 0; i < num_elements; i++){
	    printf("arr[%d] = %d\n", i, arr[i]);
    }
}

__global__ void printArray(int* arr, size_t num_elements)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("arr[%d] = %d\n", index, arr[index]);
}

int main()
{
    const size_t num_blocks = 2;
    const size_t num_threads_per_block = 3;
    size_t num_elements = num_blocks * num_threads_per_block;

    int* arr;
    size_t bytes = num_elements * sizeof(int);
    arr = (int *)malloc(bytes);
    
    for(size_t i = 0; i < num_elements; i++)   arr[i] = i;

	printf("----- CPU -----\n");
    printArrayCPU(arr, num_elements);

	printf("----- GPU -----\n");
    printArray<<<num_blocks, num_threads_per_block>>>(arr, num_elements);
    hipDeviceSynchronize();
}

/* OUTPUT

----- CPU -----
arr[0] = 0
arr[1] = 1
arr[2] = 2
arr[3] = 3
arr[4] = 4
arr[5] = 5
----- GPU -----

*/