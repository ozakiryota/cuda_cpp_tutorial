
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld()
{
    printf("Hello World!\n");
}

int main()
{
    int device_id;
    hipGetDevice(&device_id);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);

    printf("Device ID: %d\n", device_id);
    printf("Number of SMs: %d\n", props.multiProcessorCount);
    printf("Compute Capability Major: %d\n", props.major);
    printf("Compute Capability Minor: %d\n", props.minor);
    printf("Warp Size: %d\n", props.warpSize);
}

/* OUTPUT

Device ID: 0
Number of SMs: 28
Compute Capability Major: 8
Compute Capability Minor: 6
Warp Size: 32

*/