
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printIndicies()
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("index = %d\n", index);
}

int main()
{
    const size_t num_blocks = 1;
    const size_t num_threads_per_block = 1025; // > 1024

    printIndicies<<<num_blocks, num_threads_per_block>>>();
    hipDeviceSynchronize();
}

/* OUTPUT



*/