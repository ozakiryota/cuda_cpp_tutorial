
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld()
{
    printf("Hello World!\n");
}

int main()
{
    const size_t num_blocks = 1;
    const size_t num_threads_per_block = 3;

    helloWorld<<<num_blocks, num_threads_per_block>>>();
    // cudaDeviceSynchronize();
}

/* OUTPUT



*/